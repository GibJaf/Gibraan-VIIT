
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;


__global__ void  multiply(int *ad,int *bd,int *cd,int n)
{


	int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	
	int sum=0;

	
	for(int i=0;i<n;i++)

	{

	sum=sum+ad[row*n+i]*bd[i*n+col];


	}

	cd[row*n+col]=sum;


}

int main()
{

	cout<<"Enter the size"<<endl;
	int n;

	cin>>n;

	int a[n][n],b[n][n],c[n][n];

	for(int i=0;i<n;i++)
	{

		for(int j=0;j<n;j++)

		{

			a[i][j]=3+i;
			b[i][j]=2+j;

		}

	}

	int size=n*n*sizeof(int);

	int *ad,*bd,*cd;


	hipEvent_t start,end;


		


	hipMalloc(&ad,size);
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);


	hipMalloc(&bd,size);
	hipMemcpy(bd,b,size,hipMemcpyHostToDevice);


	hipMalloc(&cd,size);


	dim3 grid(n,n,n);
	dim3 block(1,1,1);


	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);
	



	multiply<<<grid,size>>>(ad,bd,cd,n);


	hipEventRecord(end);
	hipEventSynchronize(end);

	float time=0;
	
	hipEventElapsedTime(&time,start,end);

	


	hipMemcpy(c,cd,size,hipMemcpyDeviceToHost);

	for(int i=0;i<n;i++)
	{

		for(int j=0;j<n;j++)
		{

		cout<<c[i][j]<<" ";


		}
		cout<<endl;
	}


	cout<<"The time required is "<<time<<endl;

}
