
#include <hip/hip_runtime.h>
#include "stdio.h"
#define COLUMNS 5
#define ROWS 5
__global__ void add(int *a, int *b, int *c)
{
int x = blockIdx.x;
int y = blockIdx.y;
int i = (COLUMNS*y) + x;
c[i] = a[i] + b[i];
}
int main()
{
int a[ROWS][COLUMNS], b[ROWS][COLUMNS], c[ROWS][COLUMNS];
int *dev_a, *dev_b, *dev_c;
hipMalloc((void **) &dev_a, ROWS*COLUMNS*sizeof(int));
hipMalloc((void **) &dev_b, ROWS*COLUMNS*sizeof(int));
hipMalloc((void **) &dev_c, ROWS*COLUMNS*sizeof(int));
for (int y = 0; y < ROWS; y++)
// Fill Arrays
for (int x =
0; x < COLUMNS; x++)
{
a[y][x] = x;
b[y][x] = y;
}
hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dev_b, b, ROWS*COLUMNS*sizeof(int),hipMemcpyHostToDevice);
dim3 grid(COLUMNS,ROWS);
add<<<grid,1>>>(dev_a, dev_b, dev_c);
hipMemcpy(c, dev_c, ROWS*COLUMNS*sizeof(int),hipMemcpyDeviceToHost);
for (int y = 0; y < ROWS; y++)
// Output Arrays
{
for (int x = 0; x < COLUMNS; x++)
{
printf("[%d][%d]=%d ",y,x,c[y][x]);
}
printf("\n");
}
return 0;
}
