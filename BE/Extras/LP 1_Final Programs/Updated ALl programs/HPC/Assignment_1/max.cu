#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
// #define SIZE 1000
#define SIZE 10
__global__ void max(int *a , int *c)
{
int i = threadIdx.x;
// kernel function definition
// initialize i to thread ID
*c = a[0];
if(a[i] > *c)
{
*c = a[i];
}
}
int main()
{

srand(time(NULL)); 
//of the seed
//makes use of the computer's internal clock to control the choice
int a[SIZE]={12,4,7,3,9,5,11,6,1};
int c;
int *dev_a, *dev_c;
//GPU / device parameters
hipMalloc((void **) &dev_a, SIZE*sizeof(int));
//GPU from CUDA runtime API
hipMalloc((void **) &dev_c, SIZE*sizeof(int));

//assign memory to parameters on
// input the numbers
hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);
//array from CPU to GPU
max<<<1,SIZE>>>(dev_a,dev_c);
// call kernel function <<<number of blocks, number of threads
hipMemcpy(&c, dev_c, SIZE*sizeof(int),hipMemcpyDeviceToHost);
//result back from GPU to CPU
printf("\nmax = %d ",c);
//copy the
// copy thecudaFree(dev_a);
hipFree(dev_c);

return 0;
// Free the allocated memory
}
