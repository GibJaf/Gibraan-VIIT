
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

__global__ void add(int *a,int *b,int *c,int n)
{



 int id=blockIdx.x*blockDim.x+threadIdx.x;


if(id<n)
{

c[id]=b[id]+a[id];


}



}


int main()
{




cout<<"Enter the no of elements"<<endl;
int n;

cin>>n;
int a[n],b[n],c[n];

for(int i=0;i<n;i++)
{

a[i]=b[i]=i+1;

}

int *ad,*bd,*cd;

int size=n*sizeof(int);


hipEvent_t start,end;


hipEventCreate(&start);
hipEventCreate(&end);







hipMalloc(&ad,size);
hipMemcpy(ad,a,size,hipMemcpyHostToDevice);

hipMalloc(&bd,size);
hipMemcpy(bd,b,size,hipMemcpyHostToDevice);


hipMalloc(&cd,size);

dim3 grid(256,1);
dim3 block(32,1);


hipEventRecord(start);



add<<<grid,block>>>(ad,bd,cd,n);


hipEventRecord(end);
hipEventSynchronize(end);


float time=0;

hipEventElapsedTime(&time,start,end);







hipMemcpy(c,cd,size,hipMemcpyDeviceToHost);

for(int i=0;i<n;i++)
{

cout<<c[i]<<endl;


}


cout<<"The time required is"<<time<<endl;


}
