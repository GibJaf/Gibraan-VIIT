
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;


__global__ void multiply(int *ad,int *bd,int *cd,int n)
{

	int row=blockIdx.x;
	
	int sum=0;

	for(int i=0;i<n;i++)
	{


	sum=sum+ad[row*n+i]*bd[i];


	}

	cd[row]=sum;


}


int main()
{


	cout<<"Enter the size"<<endl;

	int n;
	cin>>n;

	int a[n][n],b[n], c[n];

	int size1=n*n*sizeof(int);
	int size2=n*sizeof(int);


	hipEvent_t start,end;

	hipEventCreate(&start);
	hipEventCreate(&end);


	cout<<"The intial matrices are"<<endl;


		for(int i=0;i<n;i++)
		{

			for(int j=0;j<n;j++)
			{

				a[i][j]=3+i;
				cout<<a[i][j]<<" ";
			}
			b[i]=2+i;
			cout << b[i];
			cout<<endl;
		}


	int *ad, *bd, *cd;

	hipMalloc(&ad,size1);
	hipMemcpy(ad,a,size1,hipMemcpyHostToDevice);


	hipMalloc(&bd,size2);
	hipMemcpy(bd,b,size2,hipMemcpyHostToDevice);

	hipMalloc(&cd,size2);





	dim3 grid(n,1);
	dim3 block(1,1);


	hipEventRecord(start);



	multiply<<<grid,block>>>(ad,bd,cd,n);


	hipEventRecord(end);

	hipEventSynchronize(end);


	float time=0;
	
	hipEventElapsedTime(&time,start,end);


	cout<<"The time is "<<time<<endl;




	cout<<"The multiplication is"<<endl;

	hipMemcpy(c,cd,size2,hipMemcpyDeviceToHost);

	for(int i=0;i<n;i++)
	{

		cout<<c[i]<<" ";
	}



}
