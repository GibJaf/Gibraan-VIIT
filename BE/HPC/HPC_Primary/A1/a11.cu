
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>

using namespace std;



__global__ void maxi(int *a,int *b,int n)
{


	int block=256*blockIdx.x;


	int max=0;

	for(int i=block;i<min(256+block,n);i++)
	{


		if(max<a[i])
		{

			max=a[i];
	
		}





	}
	b[blockIdx.x]=max;

}




int main()
{


	cout<<"Enter the size of array"<<endl;
	int n;
	cin>>n;
	int a[n];

	hipEvent_t start,end;

	for(int i=0;i<n;i++)
	{

		a[i]=i+1;
	}

	
	int *ad,*bd;
	int size=n*sizeof(int);
	
	hipMalloc(&ad,size);
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);

	

	int grids=ceil(n*1.0f/256.0f);
	hipMalloc(&bd,grids*sizeof(int));



	dim3 grid(grids,1);
	dim3 block(1,1);

	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);

	while(n>1)

	{

		maxi<<<grids,block>>>(ad,bd,n);
		n=ceil(n*1.0f/256.0f);
		hipMemcpy(ad,bd,n*sizeof(int),hipMemcpyDeviceToDevice);


	}

	hipEventRecord(end);
	hipEventSynchronize(end);

	float time=0;
	hipEventElapsedTime(&time,start,end);


	
	
	int ans[2];

	hipMemcpy(ans,ad,4,hipMemcpyDeviceToHost);
	
	cout<<"The maximum element is"<<ans[0]<<endl;
	
	cout<<"The time required dor it is";
	cout<<time<<endl;
	

} 
