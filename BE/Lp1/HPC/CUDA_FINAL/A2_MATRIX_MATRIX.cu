
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void matMul(int *a, int *b, int *c, int n){
	
	int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	
	int sum=0;
	
	for(int j=0;j<n;j++)
	{

	
		sum=sum+a[row*n+j]*b[j*n+col];

	}

	c[n*row+col]=sum;
}

int main(){
	
	int n;
	cin>>n;
	int *a= new int[n*n];
	int *b = new int[n*n];
	int *c = new int[n*n];
	
	for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){
			a[i*n+j] = i+1;
			b[i*n+j] = j+1;
		}
	}

	int *ad, *bd, *cd;
	hipMalloc(&ad, n*n*sizeof(int));
	hipMalloc(&bd, n*n*sizeof(int));
	hipMalloc(&cd, n*n*sizeof(int));

	hipMemcpy(ad, a, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(bd, b, n*n*sizeof(int), hipMemcpyHostToDevice);
	
	dim3 grids(n, n, 1);

	matMul<<<grids, 1>>>(ad, bd, cd, n);

	hipMemcpy(c, cd, n*n*sizeof(int), hipMemcpyDeviceToHost);
		
	for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){
			cout<<c[i*n+j]<<" ";
		}

		cout<<endl;
}

}
