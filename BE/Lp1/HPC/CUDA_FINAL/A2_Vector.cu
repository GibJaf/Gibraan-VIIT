
#include <hip/hip_runtime.h>
#include<iostream>
#include<chrono>

using namespace std;
using namespace std::chrono;

__global__ void vecAdd(int *a, int *b, int *c, int n)
{
	int block = blockIdx.x;
	if(block<n)
		c[block] = a[block]+b[block];
}

int main()
{
	int n;
	cin>>n;
	int *a=new int[n];
	int *b=new int[n];
	int *c=new int[n];
	
	for(int i=0;i<n;i++)
	{
		a[i]=i+1;
		b[i]=i+1;
	}
	
	int *ad, *bd, *cd;
	
	hipMalloc(&ad, n*sizeof(int));
	hipMemcpy(ad, a, n*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&bd, n*sizeof(int));
	hipMemcpy(bd, b, n*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&cd, n*sizeof(int));
	
	vecAdd<<<n, 1>>>(ad, bd, cd, n);
	
	hipMemcpy(c, cd, n*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i=0;i<n;i++)
		cout<<c[i]<<endl;
}
