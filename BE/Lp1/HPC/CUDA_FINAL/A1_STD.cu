
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void stdeviation(int *a, float *b, float mean, int n){
	
	int block =blockIdx.x;
	b[0] = 0.0;
	for(int i= block; i<n; i++){
		b[0] += (a[i] - mean)*(a[i] - mean);
	}
	
	b[0] = b[0]/n;

}


int main(){

	int n;
	cin>>n;
	int a[n];
	for(int i=0; i<n; i++){
		a[i] = i+1;
	}
	float mean = (n + 1)/2;
	int *ad;
	float *b;
	
	hipMalloc(&ad, n*sizeof(int));
	hipMalloc(&b, sizeof(float));

	hipMemcpy(ad, a, n*sizeof(int), hipMemcpyHostToDevice);
	
	stdeviation<<<n, 1>>> (ad, b, mean, n);

	float ans[1];
	hipMemcpy(ans, b,sizeof(float), hipMemcpyDeviceToHost);
	cout<<"Answer is: "<<sqrt(ans[0])<<endl;
}
